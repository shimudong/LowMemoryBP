#include "hip/hip_runtime.h"
#include "cutils.h"
#include "cudautils.cuh"


constexpr static int num_threads {128}; 
constexpr static int inner_repeat {8};


__device__ static constexpr float ReGELU2_a [3] {-0.04922261145617846, 1.0979632065417297, -0.048740595085551286};
__device__ static constexpr float ReGELU2_c [3] {-3.1858810036855245, -0.001178821281161997, 3.190832613414926};

__device__ static constexpr float ReSiLU2_a [3] {-0.04060357190528599, 1.080925428529668, -0.040321856624382146};
__device__ static constexpr float ReSiLU2_c [3] {-6.3050461001646445, -0.0008684942046214787, 6.325815242089708};

__device__ static constexpr float inv_sqrt2 {0.7071067811865475};


// regelu2
template <typename T>
__inline__ __device__ void regelu2_fw(T x, T& y, uint8_t& flag)
{
    float x_ {x};
    y = (1.f + ::erf(x_ * inv_sqrt2)) * x_ * 0.5f;
    flag = (x_ > ReGELU2_c[0]) + (x_ > ReGELU2_c[1]) + (x_ > ReGELU2_c[2]);
}


template <typename T>
__inline__ __device__ void regelu2_bw(T out_grad, uint8_t flag, T& in_grad)
{
    float out_grad_ {out_grad};
    in_grad = out_grad_ * (
        (flag > 0) * ReGELU2_a[0] + (flag > 1) * ReGELU2_a[1] + (flag > 2) * ReGELU2_a[2]
    );
}


// resilu2
template <typename T>
__inline__ __device__ void resilu2_fw(T x, T& y, uint8_t& flag)
{
    float x_ {x};
    y = x_ / (1.f + ::expf(-x_));
    flag = (x_ > ReSiLU2_c[0]) + (x_ > ReSiLU2_c[1]) + (x_ > ReSiLU2_c[2]);
}


template <typename T>
__inline__ __device__ void resilu2_bw(T out_grad, uint8_t flag, T& in_grad)
{
    float out_grad_ {out_grad};
    in_grad = out_grad_ * (
        (flag > 0) * ReSiLU2_a[0] + (flag > 1) * ReSiLU2_a[1] + (flag > 2) * ReSiLU2_a[2]
    );
}


template <typename T, int vec_size>
__global__ void
regelu2_fw_1d_kernel
(int64_t N, T * __restrict__ input_ptr, T * __restrict__ output_ptr, u_int8_t * __restrict__ flag_ptr)
{
    static_assert(vec_size <= 4, "vector size must be less than 4");

    const int gid_blk = num_threads * inner_repeat * vec_size * blockIdx.x;
    using vec_t = Pack<T, vec_size>;

    uint8_t flag;
    uint8_t packed_flag;
    vec_t input_vec [2]; // double buffer
    vec_t output_vec;
    int64_t gid{gid_blk + threadIdx.x * vec_size};
    input_vec[0] = *reinterpret_cast<vec_t*>(input_ptr + gid);
    int write_buffer = 1;
    #pragma unroll
    for (int r = 0; r < inner_repeat; ++r) {
        int gid_next = gid + num_threads * vec_size;
        int read_buffer = write_buffer ^ 1;
        if (gid < N) {
            if (gid_next < N)
                input_vec[write_buffer] = *reinterpret_cast<vec_t*>(input_ptr + gid_next);
            packed_flag = 0;
            #pragma unroll
            for (int k = 0; k < vec_size; ++k) {
                regelu2_fw(input_vec[read_buffer].elem[k], output_vec.elem[k], flag);
                packed_flag |= (flag <<= (2 * (k & 3)));
            }
            packflagWarpReduce<vec_size * 2>(packed_flag);
            *reinterpret_cast<vec_t*>(output_ptr + gid) = output_vec;
            if (!(gid & 3)) *(flag_ptr + gid / 4) = packed_flag;
        }
        gid = gid_next;
        write_buffer ^= 1;
    }
}


template <typename T, int grad_vec_size, int flag_vec_size>
__global__ void
regelu2_bw_1d_kernel
(int64_t N, T * __restrict__ out_grad_ptr, uint8_t * __restrict__ packed_flag_ptr, T * __restrict__ in_grad_ptr)
{
    const int gid_blk = num_threads * inner_repeat * grad_vec_size * blockIdx.x;
    using grad_vec_t = Pack<T, grad_vec_size>;
    using flag_vec_t = Pack<uint8_t, flag_vec_size>;

    grad_vec_t out_grad_vec [2]; // double buffer
    grad_vec_t in_grad_vec;

    flag_vec_t flag_vec [2]; // double buffer

    int64_t gid{gid_blk + threadIdx.x * grad_vec_size};
    out_grad_vec[0] = *reinterpret_cast<grad_vec_t*>(out_grad_ptr + gid);
    flag_vec[0] = *reinterpret_cast<flag_vec_t*>(packed_flag_ptr + gid / 4);
    int write_buffer = 1;
    #pragma unroll
    for (int r = 0; r < inner_repeat; ++r) {
        int gid_next = gid + num_threads * grad_vec_size;
        int read_buffer = write_buffer ^ 1;
        if (gid < N) {
            if (gid_next < N) {
                out_grad_vec[write_buffer] = *reinterpret_cast<grad_vec_t*>(out_grad_ptr + gid_next);
                flag_vec[write_buffer] = *reinterpret_cast<flag_vec_t*>(packed_flag_ptr + gid_next / 4);
            }
            #pragma unroll
            for (int k = 0; k < grad_vec_size; ++k) {
                int gid_k = gid + k;
                regelu2_bw(
                    out_grad_vec[read_buffer].elem[k],
                    (flag_vec[read_buffer].elem[k/4] >> ((gid_k & 3) * 2)) & 3,
                    in_grad_vec.elem[k]
                );
            }
            *reinterpret_cast<grad_vec_t*>(in_grad_ptr + gid) = in_grad_vec;
        }
        gid = gid_next;
        write_buffer ^= 1;
    }
}


template <typename T>
void regelu2_fw_1d_(int64_t N, void * input_ptr_, void * output_ptr_, void * flag_ptr_)
{
    T * input_ptr = reinterpret_cast<T*>(input_ptr_);
    T * output_ptr = reinterpret_cast<T*>(output_ptr_);
    u_int8_t * flag_ptr = reinterpret_cast<u_int8_t*>(flag_ptr_);

    dim3 blockDim(num_threads);
    if ((16 / sizeof(T) <= 4) && check_align(input_ptr, 16, N) && check_align(output_ptr, 16, N)) {
        constexpr int vec_size {16 / sizeof(T)};
        if constexpr (vec_size <= 4) {
            constexpr int blocksize = num_threads * inner_repeat * vec_size;
            dim3 gridDim{(N + blocksize - 1) / blocksize};
            regelu2_fw_1d_kernel<T, vec_size><<<gridDim, blockDim>>>
                (N, input_ptr, output_ptr, flag_ptr);
        }
    } else if ((8 / sizeof(T) <= 4) && check_align(input_ptr, 8, N) && check_align(output_ptr, 8, N)) {
        constexpr int vec_size {8 / sizeof(T)};
        if constexpr (vec_size <= 4) {
            const int vec_size {8 / sizeof(T)};
            constexpr int blocksize = num_threads * inner_repeat * vec_size;
            dim3 gridDim{(N + blocksize - 1) / blocksize};
            regelu2_fw_1d_kernel<T, vec_size><<<gridDim, blockDim>>>
                (N, input_ptr, output_ptr, flag_ptr);
        }
    } else if ((4 / sizeof(T) <= 4) && check_align(input_ptr, 4, N) && check_align(output_ptr, 4, N)) {
        constexpr int vec_size {4 / sizeof(T)};
        if constexpr (vec_size <= 4) {
            const int vec_size {4 / sizeof(T)};
            constexpr int blocksize = num_threads * inner_repeat * vec_size;
            dim3 gridDim{(N + blocksize - 1) / blocksize};
            regelu2_fw_1d_kernel<T, vec_size><<<gridDim, blockDim>>>
                (N, input_ptr, output_ptr, flag_ptr);
        }
    } else{
        constexpr int vec_size {1};
        constexpr int blocksize = num_threads * inner_repeat * vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        regelu2_fw_1d_kernel<T, vec_size><<<gridDim, blockDim>>>
            (N, input_ptr, output_ptr, flag_ptr);
    }
}


template <typename T>
void regelu2_bw_1d_(int64_t N, void * out_grad_ptr_, void * packed_flag_ptr_, void * in_grad_ptr_)
{
    T * out_grad_ptr = reinterpret_cast<T*>(out_grad_ptr_);
    u_int8_t * packed_flag_ptr = reinterpret_cast<u_int8_t*>(packed_flag_ptr_);
    T * in_grad_ptr = reinterpret_cast<T*>(in_grad_ptr_);

    dim3 blockDim(num_threads);
    if (check_align(out_grad_ptr, 16, N) && check_align(in_grad_ptr, 16, N)) {
        constexpr int grad_vec_size {16 / sizeof(T)};
        constexpr int flag_vec_size = (grad_vec_size + 4 - 1) / 4; 

        constexpr int blocksize = num_threads * inner_repeat * grad_vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        regelu2_bw_1d_kernel<T, grad_vec_size, flag_vec_size><<<gridDim, blockDim>>>
            (N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);

    } else if (check_align(out_grad_ptr, 8, N) && check_align(in_grad_ptr, 8, N)) {
        constexpr int grad_vec_size {8 / sizeof(T)};
        constexpr int flag_vec_size = (grad_vec_size + 4 - 1) / 4; 

        constexpr int blocksize = num_threads * inner_repeat * grad_vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        regelu2_bw_1d_kernel<T, grad_vec_size, flag_vec_size><<<gridDim, blockDim>>>
            (N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);

    } else if (check_align(out_grad_ptr, 4, N) && check_align(in_grad_ptr, 4, N)) {
        constexpr int grad_vec_size {4 / sizeof(T)};
        constexpr int flag_vec_size = (grad_vec_size + 4 - 1) / 4; 

        constexpr int blocksize = num_threads * inner_repeat * grad_vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        regelu2_bw_1d_kernel<T, grad_vec_size, flag_vec_size><<<gridDim, blockDim>>>
            (N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);

    } else{
        constexpr int grad_vec_size {1};
        constexpr int flag_vec_size {1}; 

        constexpr int blocksize = num_threads * inner_repeat * grad_vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        regelu2_bw_1d_kernel<T, grad_vec_size, flag_vec_size><<<gridDim, blockDim>>>
            (N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);
    }
}


template <typename T>
void regelu2_fw_1d(int64_t N, void * input_ptr, void * output_ptr, void * flag_ptr) {}

template <>
void regelu2_fw_1d<float>(int64_t N, void * input_ptr, void * output_ptr, void * flag_ptr)
{
    regelu2_fw_1d_<float>(N, input_ptr, output_ptr, flag_ptr);
}

template <>
void regelu2_fw_1d<half>(int64_t N, void * input_ptr, void * output_ptr, void * flag_ptr)
{
    regelu2_fw_1d_<half>(N, input_ptr, output_ptr, flag_ptr);
}

template <>
void regelu2_fw_1d<hip_bfloat16>(int64_t N, void * input_ptr, void * output_ptr, void * flag_ptr)
{
    regelu2_fw_1d_<hip_bfloat16>(N, input_ptr, output_ptr, flag_ptr);
}


template <typename T>
void regelu2_bw_1d(int64_t N, void * out_grad_ptr, void * packed_flag_ptr, void * in_grad_ptr) {}

template <>
void regelu2_bw_1d<float>(int64_t N, void * out_grad_ptr, void * packed_flag_ptr, void * in_grad_ptr)
{
    regelu2_bw_1d_<float>(N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);
}

template <>
void regelu2_bw_1d<half>(int64_t N, void * out_grad_ptr, void * packed_flag_ptr, void * in_grad_ptr)
{
    regelu2_bw_1d_<half>(N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);
}

template <>
void regelu2_bw_1d<hip_bfloat16>(int64_t N, void * out_grad_ptr, void * packed_flag_ptr, void * in_grad_ptr)
{
    regelu2_bw_1d_<hip_bfloat16>(N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);
}


template <typename T, int vec_size>
__global__ void
resilu2_fw_1d_kernel
(int64_t N, T * __restrict__ input_ptr, T * __restrict__ output_ptr, u_int8_t * __restrict__ flag_ptr)
{
    static_assert(vec_size <= 4, "vector size must be less than 4");

    const int gid_blk = num_threads * inner_repeat * vec_size * blockIdx.x;
    using vec_t = Pack<T, vec_size>;

    uint8_t flag;
    uint8_t packed_flag;
    vec_t input_vec [2]; // double buffer
    vec_t output_vec;
    int64_t gid{gid_blk + threadIdx.x * vec_size};
    input_vec[0] = *reinterpret_cast<vec_t*>(input_ptr + gid);
    int write_buffer = 1;
    #pragma unroll
    for (int r = 0; r < inner_repeat; ++r) {
        int gid_next = gid + num_threads * vec_size;
        int read_buffer = write_buffer ^ 1;
        if (gid < N) {
            if (gid_next < N)
                input_vec[write_buffer] = *reinterpret_cast<vec_t*>(input_ptr + gid_next);
            packed_flag = 0;
            #pragma unroll
            for (int k = 0; k < vec_size; ++k) {
                resilu2_fw(input_vec[read_buffer].elem[k], output_vec.elem[k], flag);
                packed_flag |= (flag <<= (2 * (k & 3)));
            }
            packflagWarpReduce<vec_size * 2>(packed_flag);
            *reinterpret_cast<vec_t*>(output_ptr + gid) = output_vec;
            if (!(gid & 3)) *(flag_ptr + gid / 4) = packed_flag;
        }
        gid = gid_next;
        write_buffer ^= 1;
    }
}


template <typename T, int grad_vec_size, int flag_vec_size>
__global__ void
resilu2_bw_1d_kernel
(int64_t N, T * __restrict__ out_grad_ptr, uint8_t * __restrict__ packed_flag_ptr, T * __restrict__ in_grad_ptr)
{
    const int gid_blk = num_threads * inner_repeat * grad_vec_size * blockIdx.x;
    using grad_vec_t = Pack<T, grad_vec_size>;
    using flag_vec_t = Pack<uint8_t, flag_vec_size>;

    grad_vec_t out_grad_vec [2]; // double buffer
    grad_vec_t in_grad_vec;

    flag_vec_t flag_vec [2]; // double buffer

    int64_t gid{gid_blk + threadIdx.x * grad_vec_size};
    out_grad_vec[0] = *reinterpret_cast<grad_vec_t*>(out_grad_ptr + gid);
    flag_vec[0] = *reinterpret_cast<flag_vec_t*>(packed_flag_ptr + gid / 4);
    int write_buffer = 1;
    #pragma unroll
    for (int r = 0; r < inner_repeat; ++r) {
        int gid_next = gid + num_threads * grad_vec_size;
        int read_buffer = write_buffer ^ 1;
        if (gid < N) {
            if (gid_next < N) {
                out_grad_vec[write_buffer] = *reinterpret_cast<grad_vec_t*>(out_grad_ptr + gid_next);
                flag_vec[write_buffer] = *reinterpret_cast<flag_vec_t*>(packed_flag_ptr + gid_next / 4);
            }
            #pragma unroll
            for (int k = 0; k < grad_vec_size; ++k) {
                int gid_k = gid + k;
                resilu2_bw(
                    out_grad_vec[read_buffer].elem[k],
                    (flag_vec[read_buffer].elem[k/4] >> ((gid_k & 3) * 2)) & 3,
                    in_grad_vec.elem[k]
                );
            }
            *reinterpret_cast<grad_vec_t*>(in_grad_ptr + gid) = in_grad_vec;
        }
        gid = gid_next;
        write_buffer ^= 1;
    }
}


template <typename T>
void resilu2_fw_1d_(int64_t N, void * input_ptr_, void * output_ptr_, void * flag_ptr_)
{
    T * input_ptr = reinterpret_cast<T*>(input_ptr_);
    T * output_ptr = reinterpret_cast<T*>(output_ptr_);
    u_int8_t * flag_ptr = reinterpret_cast<u_int8_t*>(flag_ptr_);

    dim3 blockDim(num_threads);
    if ((16 / sizeof(T) <= 4) && check_align(input_ptr, 16, N) && check_align(output_ptr, 16, N)) {
        constexpr int vec_size {16 / sizeof(T)};
        if constexpr (vec_size <= 4) {
            constexpr int blocksize = num_threads * inner_repeat * vec_size;
            dim3 gridDim{(N + blocksize - 1) / blocksize};
            resilu2_fw_1d_kernel<T, vec_size><<<gridDim, blockDim>>>
                (N, input_ptr, output_ptr, flag_ptr);
        }
    } else if ((8 / sizeof(T) <= 4) && check_align(input_ptr, 8, N) && check_align(output_ptr, 8, N)) {
        constexpr int vec_size {8 / sizeof(T)};
        if constexpr (vec_size <= 4) {
            const int vec_size {8 / sizeof(T)};
            constexpr int blocksize = num_threads * inner_repeat * vec_size;
            dim3 gridDim{(N + blocksize - 1) / blocksize};
            resilu2_fw_1d_kernel<T, vec_size><<<gridDim, blockDim>>>
                (N, input_ptr, output_ptr, flag_ptr);
        }
    } else if ((4 / sizeof(T) <= 4) && check_align(input_ptr, 4, N) && check_align(output_ptr, 4, N)) {
        constexpr int vec_size {4 / sizeof(T)};
        if constexpr (vec_size <= 4) {
            const int vec_size {4 / sizeof(T)};
            constexpr int blocksize = num_threads * inner_repeat * vec_size;
            dim3 gridDim{(N + blocksize - 1) / blocksize};
            resilu2_fw_1d_kernel<T, vec_size><<<gridDim, blockDim>>>
                (N, input_ptr, output_ptr, flag_ptr);
        }
    } else {
        constexpr int vec_size {1};
        constexpr int blocksize = num_threads * inner_repeat * vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        resilu2_fw_1d_kernel<T, vec_size><<<gridDim, blockDim>>>
            (N, input_ptr, output_ptr, flag_ptr);
    }
}


template <typename T>
void resilu2_bw_1d_(int64_t N, void * out_grad_ptr_, void * packed_flag_ptr_, void * in_grad_ptr_)
{
    T * out_grad_ptr = reinterpret_cast<T*>(out_grad_ptr_);
    u_int8_t * packed_flag_ptr = reinterpret_cast<u_int8_t*>(packed_flag_ptr_);
    T * in_grad_ptr = reinterpret_cast<T*>(in_grad_ptr_);

    dim3 blockDim(num_threads);
    if (check_align(out_grad_ptr, 16, N) && check_align(in_grad_ptr, 16, N)) {
        constexpr int grad_vec_size {16 / sizeof(T)};
        constexpr int flag_vec_size = (grad_vec_size + 4 - 1) / 4; 

        constexpr int blocksize = num_threads * inner_repeat * grad_vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        resilu2_bw_1d_kernel<T, grad_vec_size, flag_vec_size><<<gridDim, blockDim>>>
            (N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);

    } else if (check_align(out_grad_ptr, 8, N) && check_align(in_grad_ptr, 8, N)) {
        constexpr int grad_vec_size {8 / sizeof(T)};
        constexpr int flag_vec_size = (grad_vec_size + 4 - 1) / 4; 

        constexpr int blocksize = num_threads * inner_repeat * grad_vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        resilu2_bw_1d_kernel<T, grad_vec_size, flag_vec_size><<<gridDim, blockDim>>>
            (N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);

    } else if (check_align(out_grad_ptr, 4, N) && check_align(in_grad_ptr, 4, N)) {
        constexpr int grad_vec_size {4 / sizeof(T)};
        constexpr int flag_vec_size = (grad_vec_size + 4 - 1) / 4; 

        constexpr int blocksize = num_threads * inner_repeat * grad_vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        resilu2_bw_1d_kernel<T, grad_vec_size, flag_vec_size><<<gridDim, blockDim>>>
            (N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);

    } else{
        constexpr int grad_vec_size {1};
        constexpr int flag_vec_size {1}; 

        constexpr int blocksize = num_threads * inner_repeat * grad_vec_size;
        dim3 gridDim{(N + blocksize - 1) / blocksize};
        resilu2_bw_1d_kernel<T, grad_vec_size, flag_vec_size><<<gridDim, blockDim>>>
            (N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);
    }
}


template <typename T>
void resilu2_fw_1d(int64_t N, void * input_ptr, void * output_ptr, void * flag_ptr) {}

template <>
void resilu2_fw_1d<float>(int64_t N, void * input_ptr, void * output_ptr, void * flag_ptr)
{
    resilu2_fw_1d_<float>(N, input_ptr, output_ptr, flag_ptr);
}

template <>
void resilu2_fw_1d<half>(int64_t N, void * input_ptr, void * output_ptr, void * flag_ptr)
{
    resilu2_fw_1d_<half>(N, input_ptr, output_ptr, flag_ptr);
}

template <>
void resilu2_fw_1d<hip_bfloat16>(int64_t N, void * input_ptr, void * output_ptr, void * flag_ptr)
{
    resilu2_fw_1d_<hip_bfloat16>(N, input_ptr, output_ptr, flag_ptr);
}


template <typename T>
void resilu2_bw_1d(int64_t N, void * out_grad_ptr, void * packed_flag_ptr, void * in_grad_ptr) {}

template <>
void resilu2_bw_1d<float>(int64_t N, void * out_grad_ptr, void * packed_flag_ptr, void * in_grad_ptr)
{
    resilu2_bw_1d_<float>(N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);
}

template <>
void resilu2_bw_1d<half>(int64_t N, void * out_grad_ptr, void * packed_flag_ptr, void * in_grad_ptr)
{
    resilu2_bw_1d_<half>(N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);
}

template <>
void resilu2_bw_1d<hip_bfloat16>(int64_t N, void * out_grad_ptr, void * packed_flag_ptr, void * in_grad_ptr)
{
    resilu2_bw_1d_<hip_bfloat16>(N, out_grad_ptr, packed_flag_ptr, in_grad_ptr);
}